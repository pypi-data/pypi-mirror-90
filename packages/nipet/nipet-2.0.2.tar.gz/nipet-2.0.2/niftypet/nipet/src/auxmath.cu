#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------
CUDA C extension for Python
This extension module provides additional functionality for list-mode data
processing, converting between data structures for image reconstruction.

author: Pawel Markiewicz
Copyrights: 2018
------------------------------------------------------------------------*/


#include "auxmath.h"

#define MTHREADS 512

//=============================================================================
__global__ void var(float * M1,
	float * M2,
	float * X,
	int b,
	size_t nele) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if (idx<nele) {
		float delta = X[idx] - M1[idx];
		M1[idx] += delta / (b + 1);
		M2[idx] += delta*(X[idx] - M1[idx]);
	}
}
//=============================================================================
//=============================================================================
void var_online(float *M1, float *M2, float *X, int b, size_t nele)
{

	//do calculation of variance online using CUDA kernel <var>.
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	float *d_m1; HANDLE_ERROR(hipMalloc(&d_m1, nele * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(d_m1, M1, nele * sizeof(float), hipMemcpyHostToDevice));
	float *d_m2; HANDLE_ERROR(hipMalloc(&d_m2, nele * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(d_m2, M2, nele * sizeof(float), hipMemcpyHostToDevice));
	float *d_x; HANDLE_ERROR(hipMalloc(&d_x, nele * sizeof(float)));
	HANDLE_ERROR(hipMemcpy(d_x, X, nele * sizeof(float), hipMemcpyHostToDevice));


	int blcks = (nele + MTHREADS - 1) / MTHREADS;
	var << < blcks, MTHREADS >> >(d_m1, d_m2, d_x, b, nele);


	//copy M1 and M2 back to CPU memory
	HANDLE_ERROR(hipMemcpy(M1, d_m1, nele * sizeof(float), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(M2, d_m2, nele * sizeof(float), hipMemcpyDeviceToHost));

	hipFree(d_m1);
	hipFree(d_m2);
	hipFree(d_x);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("i> online variance calculation DONE in %fs.\n\n", 0.001*elapsedTime);
}
//=============================================================================
