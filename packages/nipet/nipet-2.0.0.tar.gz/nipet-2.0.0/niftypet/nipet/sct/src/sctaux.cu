#include "hip/hip_runtime.h"
/*------------------------------------------------------------------------
Python extension for CUDA auxiliary routines used in
voxel-driven scatter modelling (VSM)

author: Pawel Markiewicz
Copyrights: 2020
------------------------------------------------------------------------*/
#include <stdlib.h>
#include "sctaux.h"

//======================================================================
//SCATTER RESULTS PROCESSING
//======================================================================

__constant__ short c_isrng[N_SRNG];


//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
__global__ void d_sct2sn1(
	float *scts1,
	float *srslt,
	size_t offtof,
	char *xsxu,
	short *offseg,
	int NBIN)
{
	//scatter crystal index
	char ics = threadIdx.x;

	//scatter ring index
	char irs = threadIdx.y;

	//unscattered crystal index
	char icu = blockIdx.x;
	//unscattered crystal index
	char iru = blockIdx.y;



	//number of considered crystals and rings for scatter
	char nscrs = gridDim.x;
	char nsrng = gridDim.y;

	//scatter bin index for one scatter sino/plane
	short ssi = nscrs*icu + ics;
	bool pos = ((2*xsxu[ssi] - 1) * (irs - iru)) > 0;

	// ring difference index used for addressing the segment offset to obtain sino index in span-1
	unsigned short rd = __usad(c_isrng[irs], c_isrng[iru], 0);

	unsigned short rdi = (2*rd - 1*pos);
	unsigned short sni = offseg[rdi] + MIN(c_isrng[irs], c_isrng[iru]);

	atomicAdd(scts1 + sni*NBIN + ssi,
		srslt[offtof + iru*nscrs*nsrng*nscrs + icu*nsrng*nscrs + irs*nscrs + ics]);
}
//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~



//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~
__global__ void d_sct_axinterp(
	float *sct3d,
	const float *scts1,
	const int4 *sctaxR,
	const float4 *sctaxW,
	const short *sn1_sn11,
	int NBIN,
	int NSN1,
	int SPN,
	int tof_off)
{
	//scatter crystal index
	char ics = threadIdx.x;

	//unscattered crystal index (the 4s are done in the loop below)
	char icu = blockIdx.x;

	//span-1 sino index
	short sni = blockIdx.y;

	float tmp = sctaxW[sni].x * scts1[NBIN*sctaxR[sni].x + icu*blockDim.x + ics] +
				sctaxW[sni].y * scts1[NBIN*sctaxR[sni].y + icu*blockDim.x + ics] +
				sctaxW[sni].z * scts1[NBIN*sctaxR[sni].z + icu*blockDim.x + ics] +
				sctaxW[sni].w * scts1[NBIN*sctaxR[sni].w + icu*blockDim.x + ics];

	//span-1 or span-11 scatter pre-sinogram interpolation
	if (SPN == 1)
		sct3d[tof_off + sni*NBIN + icu*blockDim.x + ics] = tmp;
	else if (SPN == 11)
		if (sni<NSN1) atomicAdd(sct3d + tof_off + sn1_sn11[sni]*NBIN + icu*blockDim.x + ics, tmp);

}

//~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~


//======================================================================
float * srslt2sino(
	float *d_srslt,
	char *d_xsxu,
	scrsDEF d_scrsdef,
	int *sctaxR,
	float *sctaxW,
	short *offseg,
	short *isrng,
	short *sn1_rno,
	short *sn1_sn11,
	Cnst Cnt)
{

	//scatter pre-sino in span-1 (tmporary)
	float *d_scts1;
	HANDLE_ERROR(hipMalloc(&d_scts1, Cnt.NSN64*d_scrsdef.nscrs*d_scrsdef.nscrs * sizeof(float)));


	//axially interpolated scatter pre-sino; full span-1 without MRD limit or span-11 with MRD=60
	float *d_sct3di;
	int tbins = 0;
	if (Cnt.SPN == 1)
		tbins = Cnt.NSN64*d_scrsdef.nscrs*d_scrsdef.nscrs;
	//scatter pre-sino, span-11
	else if (Cnt.SPN == 11)
		tbins = Cnt.NSN11*d_scrsdef.nscrs*d_scrsdef.nscrs;

	HANDLE_ERROR(hipMalloc(&d_sct3di, Cnt.TOFBINN*tbins * sizeof(float)));
	HANDLE_ERROR(hipMemset(d_sct3di, 0, Cnt.TOFBINN*tbins * sizeof(float)));

	//number of all scatter estimated values (sevn) for one TOF 3D sino
	int sevn = d_scrsdef.nsrng*d_scrsdef.nscrs*d_scrsdef.nsrng*d_scrsdef.nscrs;

	//---- constants
	int4 *d_sctaxR;
	HANDLE_ERROR(hipMalloc(&d_sctaxR, Cnt.NSN64 * sizeof(int4)));
	HANDLE_ERROR(hipMemcpy(d_sctaxR, sctaxR, Cnt.NSN64 * sizeof(int4), hipMemcpyHostToDevice));

	float4 *d_sctaxW;
	HANDLE_ERROR(hipMalloc(&d_sctaxW, Cnt.NSN64 * sizeof(float4)));
	HANDLE_ERROR(hipMemcpy(d_sctaxW, sctaxW, Cnt.NSN64 * sizeof(float4), hipMemcpyHostToDevice));

	short *d_offseg;
	HANDLE_ERROR(hipMalloc(&d_offseg, (Cnt.NSEG0 + 1) * sizeof(short)));
	HANDLE_ERROR(hipMemcpy(d_offseg, offseg, (Cnt.NSEG0 + 1) * sizeof(short), hipMemcpyHostToDevice));

	if (N_SRNG != Cnt.NSRNG) printf("e> Number of scatter rings is different in definitions from Python! <<<<<<<<<<<<<<<<<<< error \n");

	//---scatter ring indices to constant memory (GPU)
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(c_isrng), isrng, Cnt.NSRNG * sizeof(short)));
	//---

	short2 *d_sn1_rno;
	HANDLE_ERROR(hipMalloc(&d_sn1_rno, Cnt.NSN1 * sizeof(short2)));
	HANDLE_ERROR(hipMemcpy(d_sn1_rno, sn1_rno, Cnt.NSN1 * sizeof(short2), hipMemcpyHostToDevice));

	short *d_sn1_sn11;
	HANDLE_ERROR(hipMalloc(&d_sn1_sn11, Cnt.NSN1 * sizeof(short)));
	HANDLE_ERROR(hipMemcpy(d_sn1_sn11, sn1_sn11, Cnt.NSN1 * sizeof(short), hipMemcpyHostToDevice));
	//----

	for (int i = 0; i<Cnt.TOFBINN; i++) {

		//offset for given TOF bin
		size_t offtof = i*sevn;

		//init to zeros
		HANDLE_ERROR(hipMemset(d_scts1, 0, Cnt.NSN64*d_scrsdef.nscrs*d_scrsdef.nscrs * sizeof(float)));


		if (Cnt.LOG <= LOGINFO) printf("i> 3D scatter results into span-1 pre-sino for TOF bin %d...", i);
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);

		//<><><><><><><><><><><><><><><><><><><><><><><><><><><><><>
		dim3 grid(d_scrsdef.nscrs, d_scrsdef.nsrng, 1);
		dim3 block(d_scrsdef.nscrs, d_scrsdef.nsrng, 1);
		d_sct2sn1 <<< grid, block >>>(d_scts1,
			d_srslt,
			offtof,
			d_xsxu,
			d_offseg,
			(int)(d_scrsdef.nscrs*d_scrsdef.nscrs));
		HANDLE_ERROR(hipGetLastError());
		//<><><><><><><><><><><><><><><><><><><><><><><><><><><><><>

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		float elapsedTime;
		hipEventElapsedTime(&elapsedTime, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);
		if (Cnt.LOG <= LOGINFO) printf("DONE in %fs.\n", 1e-3*elapsedTime);



		if (Cnt.LOG <= LOGINFO) printf("i> 3D scatter axial interpolation...");
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
		//<><><><><><><><><><><><><><><><><><><><><><><><><><><><><>
		block.x = d_scrsdef.nscrs;
		block.y = 1;
		block.z = 1;
		grid.x = d_scrsdef.nscrs;
		grid.y = Cnt.NSN1;
		grid.z = 1;
		d_sct_axinterp <<< grid, block >>>(d_sct3di,
			d_scts1,
			d_sctaxR,
			d_sctaxW,
			d_sn1_sn11,
			(int)(d_scrsdef.nscrs*d_scrsdef.nscrs),
			Cnt.NSN1,
			Cnt.SPN,
			i*tbins);
		HANDLE_ERROR(hipGetLastError());
		//<><><><><><><><><><><><><><><><><><><><><><><><><><><><><>
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsedTime, start, stop);
		hipEventDestroy(start);
		hipEventDestroy(stop);
		if (Cnt.LOG <= LOGINFO) printf("DONE in %fs.\n", 1e-3*elapsedTime);

	}

	hipFree(d_scts1);
	return d_sct3di;

	// hipFree(d_sct3di);
	// return d_scts1;

}






//===================================================================
//------ CREATE MASK BASED ON THRESHOLD (SCATTER EMISSION DATA)------------
iMSK get_imskEm(IMflt imvol, float thrshld, Cnst Cnt)
{

	// check which device is going to be used
	int dev_id;
	hipGetDevice(&dev_id);
	if (Cnt.LOG <= LOGINFO) printf("i> using CUDA device #%d\n", dev_id);

	iMSK msk;
	int nvx = 0;

	for (int i = 0; i<(SSE_IMX*SSE_IMY*SSE_IMZ); i++) {
		if (imvol.im[i]>thrshld)  nvx++;
	}
	//------------------------------------------------------------------
	//create the mask thru indexes
	int *d_i2v, *d_v2i;

#ifdef WIN32
	int *h_i2v, *h_v2i;
	HANDLE_ERROR(hipHostMalloc(&h_i2v, nvx * sizeof(int)));
	HANDLE_ERROR(hipHostMalloc(&h_v2i, SSE_IMX*SSE_IMY*SSE_IMZ * sizeof(int)));

	HANDLE_ERROR(hipMalloc(&d_i2v, nvx * sizeof(int)));
	HANDLE_ERROR(hipMalloc(&d_v2i, SSE_IMX*SSE_IMY*SSE_IMZ * sizeof(int)));

	nvx = 0;
	for (int i = 0; i<(SSE_IMX*SSE_IMY*SSE_IMZ); i++) {
		//if not in the mask then set to -1
		h_v2i[i] = 0;
		//image-based TFOV
		if (imvol.im[i]>thrshld) {
			h_i2v[nvx] = i;
			h_v2i[i] = nvx;
			nvx++;
		}
	}

	HANDLE_ERROR(hipMemcpy(d_i2v, h_i2v, nvx * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_v2i, h_v2i, SSE_IMX*SSE_IMY*SSE_IMZ * sizeof(int), hipMemcpyHostToDevice));

	HANDLE_ERROR(hipHostFree(h_i2v));
	HANDLE_ERROR(hipHostFree(h_v2i));


#else
	//printf(">>>>> NVX:%d, THRESHOLD:%f\n", nvx, thrshld);
	HANDLE_ERROR(hipMallocManaged(&d_i2v, nvx * sizeof(int)));
	HANDLE_ERROR(hipMallocManaged(&d_v2i, SSE_IMX*SSE_IMY*SSE_IMZ * sizeof(int)));

	nvx = 0;
	for (int i = 0; i<(SSE_IMX*SSE_IMY*SSE_IMZ); i++) {
		//if not in the mask then set to -1
		d_v2i[i] = 0;
		//image-based TFOV
		if (imvol.im[i]>thrshld) {
			d_i2v[nvx] = i;
			d_v2i[i] = nvx;
			nvx++;
		}
	}

#endif

	if (Cnt.LOG <= LOGINFO) printf("i> number of voxel values greater than %3.2f is %d out of %d (ratio: %3.2f)\n", thrshld, nvx, SSE_IMX*SSE_IMY*SSE_IMZ, nvx / (float)(SSE_IMX*SSE_IMY*SSE_IMZ));
	msk.nvx = nvx;
	msk.i2v = d_i2v;
	msk.v2i = d_v2i;
	return msk;
}
//===================================================================

//===================================================================
//----------- CREATE MASK BASED ON MASK PROVIDED ----------------
iMSK get_imskMu(IMflt imvol, char *msk, Cnst Cnt)
{

	// check which device is going to be used
	int dev_id;
	hipGetDevice(&dev_id);
	if (Cnt.LOG <= LOGINFO) printf("i> using CUDA device #%d\n", dev_id);

	int nvx = 0;
	for (int i = 0; i<(SS_IMX*SS_IMY*SS_IMZ); i++) {
		if (msk[i]>0)  nvx++;
	}
	//------------------------------------------------------------------
	//create the mask thru indecies
	int *d_i2v, *d_v2i;

#ifdef WIN32
	int *h_i2v, *h_v2i;
	HANDLE_ERROR(hipHostMalloc(&h_i2v, nvx * sizeof(int)));
	HANDLE_ERROR(hipHostMalloc(&h_v2i, SS_IMX*SS_IMY*SS_IMZ * sizeof(int)));

	HANDLE_ERROR(hipMalloc(&d_i2v, nvx * sizeof(int)));
	HANDLE_ERROR(hipMalloc(&d_v2i, SS_IMX*SS_IMY*SS_IMZ * sizeof(int)));

	nvx = 0;
	for (int i = 0; i<(SS_IMX*SS_IMY*SS_IMZ); i++) {
		//if not in the mask then set to -1
		h_v2i[i] = -1;
		//image-based TFOV
		if (msk[i]>0) {
			h_i2v[nvx] = i;
			h_v2i[i] = nvx;
			nvx++;
		}
	}

	HANDLE_ERROR(hipMemcpy(d_i2v, h_i2v, nvx * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_v2i, h_v2i, SS_IMX*SS_IMY*SS_IMZ * sizeof(int), hipMemcpyHostToDevice));

	HANDLE_ERROR(hipHostFree(h_i2v));
	HANDLE_ERROR(hipHostFree(h_v2i));

#else

	HANDLE_ERROR(hipMallocManaged(&d_i2v, nvx * sizeof(int)));
	HANDLE_ERROR(hipMallocManaged(&d_v2i, SS_IMX*SS_IMY*SS_IMZ * sizeof(int)));

	nvx = 0;
	for (int i = 0; i<(SS_IMX*SS_IMY*SS_IMZ); i++) {
		//if not in the mask then set to -1
		d_v2i[i] = -1;
		//image-based TFOV
		if (msk[i]>0) {
			d_i2v[nvx] = i;
			d_v2i[i] = nvx;
			nvx++;
		}
	}

#endif
	if (Cnt.LOG <= LOGINFO) printf("i> number of voxels within the mu-mask is %d out of %d (ratio: %3.2f)\n", nvx, SS_IMX*SS_IMY*SS_IMZ, nvx / (float)(SS_IMX*SS_IMY*SS_IMZ));
	iMSK mlut;
	mlut.nvx = nvx;
	mlut.i2v = d_i2v;
	mlut.v2i = d_v2i;
	return mlut;
}
